#include "hip/hip_runtime.h"
#include <calculate.h>
#include <malloc.h>
namespace zjx{
const int block_max=1024;
const int grid_max=1024;
struct grid_struct{
    int block_num;
    int grid_num;
};
struct grid_struct* compute_num(int number){

    struct grid_struct *pp = (struct grid_struct*)malloc(sizeof(struct grid_struct));
    if(number<=256){
        pp->block_num=number;
        pp->grid_num=1;
    }else{
        pp->block_num=block_max;
        pp->grid_num=number/block_max > grid_max ? grid_max:number/block_max;
    }
    return pp;
}
__global__ void addwithcuda(int64_t* pointer, int length){
    const int64_t num_threads = gridDim.x * blockDim.x;
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t i = thread_id; i < length; i += num_threads){
        *(pointer + i)=*(pointer + i) + 100;
    }
}
void add_ceshi(int* pointer, int length){
    int *pointer_cuda;
    hipMalloc((void**)&pointer_cuda,sizeof(int64_t)*length);
    hipMemcpy(pointer_cuda, pointer, sizeof(int64_t)*length, hipMemcpyHostToDevice);
    struct grid_struct *cuda_struct=compute_num(length);
    addwithcuda<<<cuda_struct->block_num, cuda_struct->grid_num >>>((int64_t*)pointer_cuda,length);
    hipMemcpy(pointer, pointer_cuda, sizeof(int64_t)*length, hipMemcpyDeviceToHost);
    hipFree(pointer_cuda);
}
}